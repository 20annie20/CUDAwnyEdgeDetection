#include "hip/hip_runtime.h"
#include "Pch.h"
#include <hip/hip_runtime.h>
#include ""
#include <math.h>
#include <stdint.h>

#include <Windows.h>

#define R 1         //filter radius
#define D R*1 + 1   //filter diameter
#define S D*D       //filter size
#define TILE 16
#define BLOCK_W 16+(2*R)
#define BLOCK_H 16+(2*R)

__global__ void kernel(const uint8_t* pixels, uint8_t* out, int width, int height, int comp)
{
    const int SOBEL_X[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
    const int SOBEL_Y[] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    int index = y * width + x;

    float LuminanceConv[3] = {0.2125f, 0.7154f, 0.0721f};

    if ((x >= 1) && (x < width - 1) && (y >= 1) && (y < height - 1)) {
        float axr = 0, axg = 0, axb = 0;
        float ayr = 0, ayg = 0, ayb = 0;
        int ix = 0;
        
        for (int i = -1; i <= 1; i++) {
            for (int j = -1; j <= 1; j++) {

                ix = pixels[3 * ((y + i) * width + x + j)];
                axr += ix * SOBEL_X[(i + 1) * 3 + j + 1];
                ayr += ix * SOBEL_Y[(i + 1) * 3 + j + 1];
                ix = pixels[3 * ((y + i) * width + x + j) + 1];
                axg += ix * SOBEL_X[(i + 1) * 3 + j + 1];
                ayg += ix * SOBEL_Y[(i + 1) * 3 + j + 1];
                ix = pixels[3 * ((y + i) * width + x + j) + 2];
                axb += ix * SOBEL_X[(i + 1) * 3 + j + 1];
                ayb += ix * SOBEL_Y[(i + 1) * 3 + j + 1];
            }
        }

        axr *= LuminanceConv[0];
        axg *= LuminanceConv[1];
        axb *= LuminanceConv[2];
        ayr *= LuminanceConv[0];
        ayg *= LuminanceConv[1];
        ayb *= LuminanceConv[2];

        out[3 * index] = max(min(sqrt((axr * axr) + (ayr * ayr)), 255.0f), 0.0f);
        out[3 * index + 1] = max(min(sqrt((axg * axg) + (ayg * ayg)), 255.0f), 0.0f);
        out[3 * index + 2] = max(min(sqrt((axb * axb) + (ayb * ayb)), 255.0f), 0.0f);
    }

}

void processImage(uint8_t* pixels, int width, int height, int comp)
{
    int size = width * height * comp * sizeof(uint8_t);

    #if 0 // breakpoint
    while (!IsDebuggerPresent())
        ;
    #endif

    void* ptr;
    hipError_t r = hipMalloc(&ptr, size);
    void* out;
    r = hipMalloc(&out, size);
    r = hipMemcpy(ptr, pixels, size, hipMemcpyHostToDevice);

    dim3 blocks(16, 16, 1);
    dim3 grid(1, 1, 1);
    grid.x = (width + blocks.x - 1) / blocks.x;
    grid.y = (height + blocks.y - 1) / blocks.y;

    kernel<<<grid, blocks>>>((uint8_t*)ptr, (uint8_t*)out, width, height, comp);

    //r = hipDeviceSynchronize();
    r = hipMemcpy(pixels, out, size, hipMemcpyDeviceToHost);
    r = hipFree(ptr);
    r = hipFree(out);
}
